//gcc breaker.c -o breaker -lcrypt && ./breaker



#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <math.h>


#define MAX_SIZE 256

//Performance:
#define NB_BLOCKS 64
#define NB_THREADS_PER_BLOCK 128
#define MAX_ITER 200

__device__ void increment_brute_force(unsigned char* tab, int length, int loop_size, int index)
{
    int j;
    for(j=0; j<length; j++)
    {
        int add = index + tab[j];
        tab[j] = add % loop_size;
        index = add / loop_size;
    }
}

__device__ int get_keypair(unsigned char* tab, int length, int first_char, int last_char){

	int sum=0;
	int pow=1;

	int i=0;
	for(i=0; i<length; i++)
	{
		sum+=(tab[i]+first_char)*pow;
		pow*=(last_char - first_char);	
	}

	return sum;
}

__global__ void breaker(int numThreads, int length, int first_char, int last_char, int crypt, unsigned char* d_brute, unsigned char* d_password)
{
	//compute our index number
    uint idx = (blockIdx.x*blockDim.x + threadIdx.x);

	unsigned char tab[MAX_SIZE];
    int loop_size = last_char - first_char;
    int current_keypair;
	int i,j;

	for(j = 0; j < length; j++)
	{
		tab[j] = d_brute[j];
	}
	
	increment_brute_force(tab, length, loop_size, idx);
	for(i = 0; i < MAX_ITER; i++)
	{
        //get keypair
		current_keypair = get_keypair(tab, length, first_char, last_char);

		//compare keypairs
		if( crypt == current_keypair )
		{
			//save correct password
			for(j= 0; j < MAX_SIZE; j++)
			{
				d_password[j] = tab[j]+first_char;
			}
            d_password[length] = 0;
		}
		increment_brute_force(tab, length, loop_size, numThreads);
    }
}

__host__ bool increment_brute_force_host(unsigned char* tab, int length, int loop_size, int index)
{
    int j;
    for(j=0; j<length; j++)
    {
        int add = index + tab[j];
        tab[j] = add % loop_size;
        index = add / loop_size;
    }

    return index != 0; 
}

__host__ int get_keypair_host(unsigned char* tab, int length, int first_char, int last_char){

	int sum=0;
	int pow=1;

	int i=0;
	for(i=0; i<length; i++)
	{
		sum+=tab[i]*pow;
		pow*=(last_char - first_char);	
	}

	return sum;
}

int main( int argc, char** argv ) {
    char *password; 
	int first_char, last_char, length;
	float t1, t2; 

	int numThreads = NB_BLOCKS * NB_THREADS_PER_BLOCK;
	int k = 0;

	unsigned char h_brute[MAX_SIZE];
	unsigned char h_password[MAX_SIZE];
    unsigned char *d_brute;
    unsigned char *d_password;
	int crypted;
    bool end = false;
	
    if( argc == 1 ) {
		password = "A$4c";
		first_char = 32;
		last_char = 126;
        length = 4;
		/* ---ASCII values---
		 * special characters: 	32 to 47
		 * numbers: 		48 to 57
		 * special characters: 	58 to 64
		 * letters uppercase: 	65 to 90
		 * special characters: 	91 to 96
		 * letters lowercase: 	97 to 122
		 * special characters: 	123 to 126
		 * */
	} else if( argc == 4 ) {
        length = strlen(argv[1]);
        if(length > MAX_SIZE)
        {
            printf( "Too long password, shortened to MAX_SIZE %d.\n", MAX_SIZE);
            length = MAX_SIZE;
        }
		password = argv[1];
		first_char = atoi( argv[2] );
		last_char = atoi( argv[3] );
	} else {
		printf("usage: breaker <password> <first_ch> <last_ch>\n");
		printf("default: breaker A$4c 32 126\n");
		printf("exemple to break the binary password 1101000:\n");
		printf( "breaker 1101000 48 49\n" );
		exit( 0 );
	}

	crypted = get_keypair_host((unsigned char*)password, length, first_char, last_char);

    for(k = 0; k < MAX_SIZE; k++)
    {
        h_brute[k] = 0;
        h_password[k] = 0;
    }

	printf( "*running parameters*\n" );
	printf( " -password length:\t%d digits\n", length );
	printf( " -digits:\t\tfrom -%c- to -%c-\n", first_char, last_char );
	printf(	" -crypted to break:\t%d\n", crypted );

	t1 = clock();
    
    //d_password : the table used by the kernel to store retrieved password 
    hipMalloc((void**)&d_password, MAX_SIZE*sizeof(unsigned char));
    hipMalloc((void**)&d_brute, MAX_SIZE*sizeof(unsigned char));

    hipMemcpy(d_password, h_password, MAX_SIZE, hipMemcpyHostToDevice);

    //
	do{
		hipMemcpy(d_brute, h_brute, MAX_SIZE, hipMemcpyHostToDevice);
		
		dim3 dimGrid(NB_BLOCKS);
        dim3 dimBlock(NB_THREADS_PER_BLOCK);

        //Start breaker loop : each kernel will perform MAX_ITER password checks
		breaker<<<dimGrid, dimBlock>>>(numThreads, length, first_char, last_char, crypted, d_brute, d_password);

		//look for a matching password
		hipMemcpy(&h_password, d_password, MAX_SIZE, hipMemcpyDeviceToHost);

		if(h_password[0] != 0)
		{
			printf("\n\nFOUND password: ");
            k = 0;
			while(h_password[k] != 0)
			{
				printf("%c", h_password[k]);
#if EXHAUSTIVE_SEARCH                
                //The following can be performed in case we want to retrieve all matching passwords
                h_password[k] = 0;
#endif
				k++;
			}

			printf("\n");

#if EXHAUSTIVE_SEARCH
            //If exhaustive search, reinitialize password tables
            cudaMemcpy(d_password, h_password, MAX_SIZE, cudaMemcpyHostToDevice);
#else
            //If not exhaustive search, stop research at first match
            goto exit;
#endif
		}

        //Increment the table used for brute force, as numThreads*MAX_ITER have already been checked
		end = increment_brute_force_host(h_brute, length, last_char-first_char, numThreads * MAX_ITER);

	} while(!end);

exit:
    t2 = clock();

	float period = (t2-t1)/CLOCKS_PER_SEC;
	if( period < 60 )
		printf( "time: %.1fs \n", period );
	else
		printf( "time: %.1fmin \n", period/60 );

	return EXIT_SUCCESS;
}
