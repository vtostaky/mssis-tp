
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>


__global__ void reduce_kernel(float *in, float *out, int ntot)
{
    int i = blockIdx.x;
    int j = i * blockDim.x + threadIdx.x;
 
    if(j < ntot)
        out[i] += in[j];
}

__host__ void init_vec(float *h_in, int ntot)
{
    for(int i = 0 ; i < ntot ; i++)
    {
			h_in[i] = sinf(float(i));
    }
}

__host__ void verif(float sum, float *h_in, int ntot)
{
    float sum_res = 0.;
    for(int i = 0 ; i < ntot ; i++)
    {
			sum_res += h_in[i];
    }
    float err = fabsf((sum - sum_res)/sum);
    printf("GPU sum : %.4e\n", sum);
    printf("CPU sum : %.4e\n", sum_res);
    if (err < 1.e-4)
    {
			printf("TEST PASSED (err %.4e < 1.e-4).\n", err);
    }
 	  else
    {
			printf("TEST FAILED (err %.4e > 1.e-4).\n", err);
    }
}

int main(int argc, char **argv)
{
    float sum;
    int nthreads, nblocks, ntot;
    int i, j;

    nthreads = 128;
    ntot = atoi(argv[1]);
    nblocks = (ntot + nthreads - 1) / nthreads;

    printf("Ntot     : %d\n", ntot);
    printf("nthreads : %d\n", nthreads);
    printf("nblocks  : %d\n", nblocks);

    float *d_sum, *d_bl, *d_in, *h_in, *h_bl;

    h_in = (float*)malloc(ntot*sizeof(float));
    h_bl = (float*)malloc(nblocks*sizeof(float));

    hipMalloc((void**)&d_sum, sizeof(float));
    hipMalloc((void**)&d_bl, nblocks*sizeof(float));
    hipMalloc((void**)&d_in, ntot*sizeof(float));

    init_vec(h_in, ntot);
    for(j = 0; j < ntot; j++)
        printf("|%f",h_in[j]);
    hipMemcpy(d_in, h_in, ntot*sizeof(float), hipMemcpyHostToDevice);

    // TODO : la réduction de d_in a lieu ici, le resultat est obtenu dans *d_sum
    dim3 dimGrid(nblocks,1,1);
    dim3 dimBlock(nthreads,1,1);
    reduce_kernel<<<dimGrid, dimBlock>>>(d_in, d_bl, ntot);

    hipMemcpy(h_bl, d_bl, sizeof(float), hipMemcpyDeviceToHost);

    for(i = 0; i < nblocks; i++)
        printf("Block %d : ",i);
        for(j = 0; j < ntot / nblocks; j++)
            printf("|%f",h_in[i*nblocks+j]);
        printf(" - result %f\n",h_bl[i]);
    
    //cudaMemcpy(&sum, d_sum, sizeof(float), cudaMemcpyDeviceToHost);
    
    //verif(sum, h_in, ntot);

    hipFree(d_sum);
    hipFree(d_bl);
    hipFree(d_in);
    free(h_in);

    return 0;
}

