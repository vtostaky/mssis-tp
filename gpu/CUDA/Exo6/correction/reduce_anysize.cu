
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>


__global__ void reduce_kernel(float *in, float *out, int ntot)
{
    // TODO : coder ici
     int nthreads = 1;
     int totthreads = blockDim.x;
     int test = 2;
     int index = blockIdx.x * blockDim.x + threadIdx.x;
     while(nthreads!=totthreads)
     {
	if((index % test == 0) && (index + nthreads < ntot))
        {
          in[index] = in[index] + in[index + nthreads];  
        }
        __syncthreads();
        nthreads = nthreads * 2;
        test = test * 2;
     }
  
   out[blockIdx.x] = in[blockIdx.x * blockDim.x];
}

__host__ void init_vec(float *h_in, int ntot)
{
    for(int i = 0 ; i < ntot ; i++)
    {
			h_in[i] = sinf(float(i));
    }
}

__host__ void verif(float sum, float *h_in, int ntot)
{
    float sum_res = 0.;
    for(int i = 0 ; i < ntot ; i++)
    {
			sum_res += h_in[i];
    }
    float err = fabsf((sum - sum_res)/sum);
    printf("GPU sum : %.4e\n", sum);
    printf("CPU sum : %.4e\n", sum_res);
    if (err < 1.e-4)
    {
			printf("TEST PASSED (err %.4e < 1.e-4).\n", err);
    }
 	  else
    {
			printf("TEST FAILED (err %.4e > 1.e-4).\n", err);
    }
}

int main(int argc, char **argv)
{
    float sum;
    int nthreads, nblocks, ntot;

    nthreads = 128;
    ntot = atoi(argv[1]);
    nblocks = (ntot + nthreads - 1) / nthreads;

    printf("Ntot     : %d\n", ntot);
    printf("nthreads : %d\n", nthreads);
    printf("nblocks  : %d\n", nblocks);

    float *d_sum, *d_bl, *d_in, *h_in;

    h_in = (float*)malloc(ntot*sizeof(float));

    hipMalloc((void**)&d_sum, sizeof(float));
    hipMalloc((void**)&d_bl, nblocks*sizeof(float));
    hipMalloc((void**)&d_in, ntot*sizeof(float));

    init_vec(h_in, ntot);
    hipMemcpy(d_in, h_in, ntot*sizeof(float), hipMemcpyHostToDevice);

    // TODO : la réduction de d_in a lieu ici, le resultat est obtenu dans *d_sum
    dim3 block(nblocks,1,1);
    dim3 thread(nthreads,1,1);
    dim3 unique(1,1,1);

    reduce_kernel<<<block, thread>>>(d_in, d_bl, ntot);
    
    reduce_kernel<<<unique,block>>>(d_bl, d_sum, ntot);

    hipMemcpy(&sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
    
    verif(sum, h_in, ntot);

    hipFree(d_sum);
    hipFree(d_bl);
    hipFree(d_in);
    free(h_in);

    return 0;
}

