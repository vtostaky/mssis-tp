#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define SIZE 102400
#define MOD 102399
#define STEP 1

/* ARRAY A INITIALIZER */
void init_a(int * a)
{
    int i;
    for(i=0; i<SIZE; i++)
    {
        a[i] = 1;
    }
}

/* ARRAY B INITIALIZER */
void init_b(int * b)
{
	int i, j;

	j=0;

	for(i=0; i<SIZE-1; i++)
	{
		b[j] = i;
		j = (j+STEP)%MOD;
	}	

    b[SIZE-1] = SIZE-1;
}

/* CHECKING A VALUES */
int check_a(int * a)
{
    int i;
    int correct = 1;
	for(i=0; i<SIZE; i++)
	{
		if(a[i] != (i+1)) 
		{
         
			correct = 0;
		} 
	}	

    return correct;
}


/* CUDA FUNCTION */
__global__ void mykernel(int * a, int * b, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    for(; i < N; i+= gridDim.x * blockDim.x)
    {
        a[b[i]] += b[i];
    }
}


int main(int argc, char * argv[])
{

	int * a = (int *)malloc(sizeof(int)*SIZE);
	int * b = (int *)malloc(sizeof(int)*SIZE);
    int *d_a, *d_b;

    init_a(a);
	init_b(b);

	hipMalloc((void**)&d_a,sizeof(int)*SIZE);
	hipMalloc((void**)&d_b,sizeof(int)*SIZE);
    
    hipMemcpy(d_a, a, sizeof(int)*SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(int)*SIZE, hipMemcpyHostToDevice);

	dim3 nBlocks;
	dim3 nThperBlock;

	nBlocks.x = 16;
	
	nThperBlock.x = 1024;

    struct timeval tv_start, tv_stop;
    gettimeofday(&tv_start, 0);

	mykernel<<< nBlocks , nThperBlock >>>(d_a, d_b, SIZE);
    hipDeviceSynchronize();
    gettimeofday(&tv_stop, 0);

    double time = (1000000.0*(tv_stop.tv_sec-tv_start.tv_sec) + tv_stop.tv_usec-tv_start.tv_usec)/1000.0;

    printf("Time to generate:  %3.6fs \n",time);
    
    hipMemcpy(a, d_a, sizeof(int)*SIZE, hipMemcpyDeviceToHost);

	int correct = check_a(a);;
	
	if(0 == correct)
	{
		printf("\n\n ******************** \n ***/!\\ ERROR /!\\ *** \n ******************** \n\n");
	}
	else
	{
		printf("\n\n ******************** \n ***** SUCCESS! ***** \n ******************** \n\n");
	}

    hipFree(d_a);
    hipFree(d_b);

	return 1;
}
